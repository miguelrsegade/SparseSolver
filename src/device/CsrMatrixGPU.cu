#include "CsrMatrixGPU.hpp"
#include "utils.h"
#include <cassert>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_kernels_csrmatrix.hpp"

const int BLOCKSIZE = 4;

CsrMatrixGPU::CsrMatrixGPU ( const CsrMatrix& hostMatrix) 
    : Matrix(hostMatrix)
{
    checkCudaErrors(hipMalloc(&d_mData, mNnz*sizeof(double)));
    checkCudaErrors(hipMalloc(&d_mColInd, mNnz*sizeof(int)));
    checkCudaErrors(hipMalloc(&d_mRowPtr, (mNrows+1)*sizeof(int)));

    checkCudaErrors(hipMemcpy( d_mData, hostMatrix.mData,
                            mNnz*sizeof(double),
                            hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( d_mColInd, hostMatrix.mColInd,
                            mNnz*sizeof(int),
                            hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( d_mRowPtr, hostMatrix.mRowPtr,
                            (mNrows+1)*sizeof(int),
                            hipMemcpyHostToDevice));


}

CsrMatrixGPU::CsrMatrixGPU (int n, int nnz) : Matrix(n, nnz)
{

    checkCudaErrors(hipMalloc(&d_mData, mNnz*sizeof(double)));
    checkCudaErrors(hipMalloc(&d_mColInd, mNnz*sizeof(int)));
    checkCudaErrors(hipMalloc(&d_mRowPtr, (mNrows+1)*sizeof(int)));

}

CsrMatrixGPU::~CsrMatrixGPU()
{
   checkCudaErrors(hipFree(d_mData));
}
void CsrMatrixGPU::copyToHost(CsrMatrix& hostMatrix)
{
    checkCudaErrors(hipMemcpy ( hostMatrix.mData, d_mData, 
                            mNnz*sizeof(double),
                            hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy ( hostMatrix.mColInd, d_mColInd,
                            mNnz*sizeof(int),
                            hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy ( hostMatrix.mRowPtr, d_mRowPtr,
                            (mNrows+1)*sizeof(int),
                            hipMemcpyDeviceToHost));

}

CsrMatrixGPU& CsrMatrixGPU::operator=(const CsrMatrixGPU& otherMatrix)
{
    assert(mNrows == otherMatrix.mNrows &&
            mNcols == otherMatrix.mNrows);

    checkCudaErrors(hipMemcpy ( d_mData, otherMatrix.d_mData,
                            mNnz*sizeof(double),
                            hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy ( d_mColInd, otherMatrix.d_mColInd,
                            mNnz*sizeof(int),
                            hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy ( d_mRowPtr, otherMatrix.d_mRowPtr,
                            (mNrows+1)*sizeof(int),
                            hipMemcpyDeviceToDevice));
    return *this;
}

// Sum of two matrices
// Only works for same profile matrices
CsrMatrixGPU CsrMatrixGPU::operator+(const CsrMatrixGPU& m1) const
{
    CsrMatrixGPU result(mNrows, mNnz);
    dim3 BlockSize(BLOCKSIZE);
    dim3 GridSize(mNnz / BLOCKSIZE +1 );
    kernel_csrmatrix_add <<<GridSize, BlockSize>>> (mNnz,
                        d_mData, m1.d_mData, result.d_mData);

    // ColInd and RowPtr are equal 
    checkCudaErrors(hipMemcpy ( result.d_mColInd, d_mColInd, 
                            mNnz*sizeof(int),
                            hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy ( result.d_mRowPtr, d_mRowPtr, 
                            (mNrows+1)*sizeof(int),
                            hipMemcpyDeviceToDevice));
    return result;
    
}

// Diferences of two matrices
// Only works for same profile matrices
CsrMatrixGPU CsrMatrixGPU::operator-(const CsrMatrixGPU& m1) const
{
    CsrMatrixGPU result(mNrows, mNnz);
    dim3 BlockSize(BLOCKSIZE);
    dim3 GridSize(mNnz / BLOCKSIZE +1 );
    kernel_csrmatrix_substract <<<GridSize, BlockSize>>> (mNnz,
                        d_mData, m1.d_mData, result.d_mData);

    // ColInd and RowPtr are equal 
    checkCudaErrors(hipMemcpy ( result.d_mColInd, d_mColInd, 
                            mNnz*sizeof(int),
                            hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy ( result.d_mRowPtr, d_mRowPtr, 
                            (mNrows+1)*sizeof(int),
                            hipMemcpyDeviceToDevice));
    return result;
    
}

// Only works for same profile matrices
CsrMatrixGPU CsrMatrixGPU::operator*(const double a) const
{
    CsrMatrixGPU result(mNrows, mNnz);
    dim3 BlockSize(BLOCKSIZE);
    dim3 GridSize(mNnz / BLOCKSIZE +1 );
    kernel_csrmatrix_scalarmultiply <<<GridSize, BlockSize>>> (mNnz, a,
                        d_mData, result.d_mData);

    // ColInd and RowPtr are equal 
    checkCudaErrors(hipMemcpy ( result.d_mColInd, d_mColInd, 
                            mNnz*sizeof(int),
                            hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy ( result.d_mRowPtr, d_mRowPtr,
                            (mNrows+1)*sizeof(int),
                            hipMemcpyDeviceToDevice));
    return result;
    
}
VectorGPU CsrMatrixGPU::operator*(const VectorGPU& v)
{
    VectorGPU result(mNrows);
    dim3 BlockSize(BLOCKSIZE);
    dim3 GridSize(mNnz / BLOCKSIZE +1 );
    kernel_csrmatrix_matrixvector <<< GridSize, BlockSize >>> (mNrows, mNnz,
            d_mData, d_mColInd, d_mRowPtr, v.d_mData, result.d_mData);

    return result;
}
